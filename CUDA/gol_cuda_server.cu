

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <math.h>


#define ALIVE 1
#define DEAD 0

void free_gen(unsigned int *gen){

        free(gen);
}

void swap(unsigned int **old, unsigned int ** new_) {

    unsigned int *temp = *old;

    *old = *new_;
    *new_ = temp;

}

// Allocate a matrix so as to have elements contiguos in memory
unsigned int * allocate_empty_gen(int rows, int cols)
{

        //allocate memory for an array of pointers and then allocate memory for every row
        unsigned int *gen = (unsigned int *)malloc(rows*cols* sizeof(unsigned int));

        return gen;

}

void show(unsigned int *curr_gen, int nRows, int nCols) {

        int i,j;

        printf("\033[H");
        for (i = 0; i < nRows; i++) {
                for (j = 0; j < nCols; j++) printf(curr_gen[i* nCols + j] ? "\033[07m  \033[m" : "  ");
                printf("\033[E");
        }
        fflush(stdout);
        usleep(200000);
}


void printbig(unsigned int *curr_gen, int nRows, int nCols, int z) {

        int i,j;

        FILE *f;

        if(z == 0) f = fopen("glife_cuda.txt", "w" );
        else f = fopen("glife_cuda.txt", "a" );

        for (i = 0; i < nRows; i++) {
                for (j = 0; j < nCols; j++) fprintf (f,"%c", curr_gen[i* nCols + j] ? 'x' : ' ');
                fprintf(f,"\n");
        }

        //separate fisrt evolution from last
        if( z == 0)
                fprintf(f,"\n\n\n\n\n\n ******************************************************************************************** \n\n\n\n\n\n");

        fflush(f);
        fclose(f);
}

// compute the elapsed wall-clock time between two time intervals. in ms
double elapsed_wtime(struct timeval start, struct timeval end) {

    return (double)((end.tv_sec * 1000000 + end.tv_usec) -
                       (start.tv_sec * 1000000 + start.tv_usec))/1000;


}


void writeFile(char* fileName, int w, int h, int z, bool first, double time , int n_core){
    FILE *f;


    if(first)   f = fopen(fileName, "w" );
    else f = fopen(fileName, "a" );

    if(first) fprintf(f,"%d-%d-%d,\n",w , h, z);

    // write file
    fprintf(f,"%d,%f",n_core , time);

    fprintf(f,"\n");
        fflush(f);
        fclose(f);

}

__device__ int compute_neighbor(int i, int j, int nRows, int nCols){

        // Guarda come vengono gestiti i bordi nell'originale
        int x = i % nRows;
        int y = j % nCols;
        return  x * nCols + y;
}


//int tid = threadIdx.x + blockIdx.x * blockDim.x;
// number of threds: ( N + number_thred_per_block) / number_thred_per_block

//MEMORY COALESCED ACCESS --> improve performace taking per rows

/*A 2D matrix is stored as 1D in memory:
        - in row-major layout, the element(x,y) ca be adressed as x*width+ y
        - A gen is composed by block, each block is composed by threads. All threads in same block have same block index.
        - to esure that  the extra threads do not do any work --> if(row<width && col<width) { --> written in the kernel
                                                                                                                                then do work
                                                                                                                          }
*/

/*
* a cell is born, if it has exactly three neighbours
* a cell dies of loneliness, if it has less than two neighbours
* a cell dies of overcrowding, if it has more than three neighbours
* a cell survives to the next generation, if it does not die of loneliness
* or overcrowding
*/
__global__ void cuda_evolve(unsigned int *curr_gen, unsigned int *next_gen, int nRows, int nCols, int block_size){


        const int bx = blockIdx.x, by = blockIdx.y;
        const int tx = threadIdx.x, ty = threadIdx.y;

        const int i = by * blockDim.y + ty;
        const int j = bx * blockDim.x + tx;

        //to esure that  the extra threads do not do any work
        if( !( i < nRows && j < nCols) ) return;

        int nAliveNeig = 0;

        // index --> i * nCols + j

        //compute the neighbors indexes
        int top_left =    compute_neighbor(i-1, j-1, nRows, nCols);
        int left =        compute_neighbor(i, j-1, nRows, nCols);
        int bottom_left = compute_neighbor(i+1, j-1, nRows, nCols);
        int top =         compute_neighbor(i-1, j, nRows, nCols);
        int top_right =   compute_neighbor(i-1, j+1, nRows, nCols);
        int right =       compute_neighbor(i, j+1, nRows, nCols);
        int bottom_right= compute_neighbor(i+1, j+1, nRows, nCols);
        int bottom =      compute_neighbor(i+1, j, nRows, nCols);

        //calculate how many neighbors around 3x3 are alive
        nAliveNeig = curr_gen[top_left] + curr_gen[left] + curr_gen[bottom_left]
                     +  curr_gen[top] + curr_gen[top_right] + curr_gen[right]
                     + curr_gen[bottom_right] + curr_gen[bottom];

        // store computation in next_gen
        next_gen[ i * nCols + j] = ( nAliveNeig == 3 || (nAliveNeig == 2 && curr_gen[ i * nCols + j]));

}



void game(int nRows, int nCols, int timestep, int block_size ){

        int z, x, y;
        struct timeval start, end;
        double tot_time = 0.;

        // allocation in CPU and initialization
        unsigned int * curr_gen = allocate_empty_gen(nRows, nCols);
        unsigned int * next_gen = allocate_empty_gen(nRows, nCols);


        //srand(10);
        for (x = 0; x < nRows; x++) for (y = 0; y < nCols; y++) curr_gen[x * nCols + y] = rand() < RAND_MAX / 10 ? ALIVE : DEAD;

        // allocation in GPU
        size_t gen_size = nRows * nCols * sizeof(unsigned int);

        unsigned int *cuda_curr_gen;
        unsigned int *cuda_next_gen;

        hipMalloc((void ** ) &cuda_curr_gen, gen_size );
        hipMalloc((void ** ) &cuda_next_gen, gen_size );

        // copy matrix from the host (CPU) to the device (GPU)
        hipMemcpy(cuda_curr_gen, curr_gen, gen_size, hipMemcpyHostToDevice);

        // make a 2D grid of threads, with  block_size threads in total.
        int grid_threads = (int) sqrt(block_size);
        dim3 n_threads(grid_threads, grid_threads);

        // how many blocks from the grid dim
        dim3 n_blocks;
        n_blocks.x = ( nCols + n_threads.x - 1)/n_threads.x;
        n_blocks.y = ( nRows + n_threads.y - 1)/n_threads.y;

        if( nCols > 1000 ) printbig(curr_gen, nRows, nCols, 0);

        for(z=0; z < timestep; z++){

                if(nCols <= 1000){
                        hipMemcpy(curr_gen, cuda_curr_gen, gen_size, hipMemcpyDeviceToHost);
                        show(curr_gen, nRows, nCols);
                }

                // get starting time at iteration z
                gettimeofday(&start, NULL);


                // Call Kernel on GPU
                cuda_evolve<<<n_blocks, n_threads>>>(cuda_curr_gen, cuda_next_gen, nRows, nCols, block_size);
                hipDeviceSynchronize();

                //swap cur_gen and next_gen when all the threads are done
                swap(&cuda_curr_gen, &cuda_next_gen);


                // get ending time of iteration z
                gettimeofday(&end, NULL);

                // sum up the total time execution
                tot_time += (double) elapsed_wtime(start, end);

                if (nCols > 1000)
                        printf("Iteration %d is : %f ms\n", z, (double) elapsed_wtime(start, end));

        }


        if( nCols > 1000 ){
                hipMemcpy(curr_gen, cuda_curr_gen, gen_size, hipMemcpyDeviceToHost);
                printbig(curr_gen, nRows, nCols, z);
        }

        // Save time execution
        char *fileName = (char*)malloc(50 * sizeof(char));
        sprintf(fileName, "Results/CUDA-%d-%d-%d.txt", nCols, nRows, timestep);

        writeFile(fileName, nCols, nRows, timestep, (block_size==32), tot_time, block_size);
        free(fileName);

        //free GPU memory
        hipFree(cuda_curr_gen);
        hipFree(cuda_next_gen);

        //free CPU memory
        free_gen(curr_gen);
        free_gen(next_gen);

}




int main(int c, char **v) {
        int w = 0, h = 0, t = 0, block_size = 32;

        if (c > 1) w = atoi(v[1]);
        if (c > 2) h = atoi(v[2]);
        if (c > 3) t = atoi(v[3]);
        if (c > 4) block_size = atoi(v[4]);

        if (w <= 0) w = 30;
        if (h <= 0) h = 30;
        if (t <= 0) t = 100;
        if (block_size < 32) block_size = 32; // number of threads per block

        game(w, h, t, block_size);
}




















