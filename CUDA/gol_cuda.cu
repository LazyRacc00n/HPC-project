#include "hip/hip_runtime.h"
//WEWE
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h> //ONLY UNIX SYSTEM TODO: uncomment on the cluster

#define ALIVE 1
#define DEAD 0

void free_gen(unsigned int **gen){

	free(gen[0]);
	free(gen);

}

void swap(unsigned int ***old, unsigned int ***new) {
	
    unsigned int **temp = *old;

    *old = *new;
    *new = temp;

}



// Allocate a matrix so as to have elements contiguos in memory
unsigned int ** allocate_empty_gen(int rows, int cols)
{

	int i;
	//allocate memory for an array of pointers and then allocate memory for every row
	unsigned int *gen = (unsigned int *)malloc(rows*cols* sizeof(unsigned int));
	unsigned int **array = (unsigned int **)malloc(rows*sizeof(unsigned int*));
	for (i = 0; i < rows; i++)
		array[i] = &(gen[cols*i]);

	return array;
}

//TODO: TO MODIFY
void show(void *u, int w, int h) {
	int x,y;
	int (*univ)[w] = u;
	printf("\033[H");
	for (y = 0; y < h; y++) {
		for (x = 0; x < w; x++) printf(univ[y][x] ? "\033[07m  \033[m" : "  ");
		printf("\033[E");
	}
	fflush(stdout);
	//usleep(200000);
}


//TODO: TO MODIFY
void printbig(void *u, int w, int h, int z) {
	int x,y;
	int (*univ)[w] = u;
	
	FILE *f;
	
	if(z == 0) f = fopen("glife.txt", "w" );
	else f = fopen("glife.txt", "a" );
	
	for (y = 0; y < h; y++) {
		for (x = 0; x < w; x++) fprintf (f,"%c", univ[y][x] ? 'x' : ' ');
		fprintf(f,"\n");
	}
	fprintf(f,"\n\n\n\n\n\n ******************************************************************************************** \n\n\n\n\n\n");
	fflush(f);
	fclose(f);
}


// compute neighbors in around 3x3
int compute_neighbor(int i, int j, int nRows, int nCols){

	// Guarda come vengono gestiti i bordi nell'originale	
	int x = (i + nRows) % nRows;
	int y = (j + nCols) % nCols;
	return  x * nCols + y;
}


//int tid = threadIdx.x + blockIdx.x * blockDim.x;
// number of threds: ( N + number_thred_per_block) / number_thred_per_block

//MEMORY COALESCED ACCESS --> improve performaze taking per rows

/*A 2D matrix is stored as 1D in memory:
	- in row-major layout, the element(x,y) ca be adressed as x*width+ y
	- A gen is composed by block, each block is composed by threads. All threads in same block have same block index.
	- to esure that  the extra threads do not do any work --> if(row<width && col<width) { --> written in the kernel
																then do work
															  }
*/
__global__ void cuda_evolve(unsigned int *curr_gen, unsigned int *next_gen, int nRows, int nCols, int block_size){

	const int bx = blockIdx.x, by = blockIdx.y;
    const int tx = threadIdx.x, ty = threadIdx.y;
    
	//TODO: capire se è da usaere blockDim.y o va bene 
	const int i = by * blockDim.y + ty; 
    const int j = bx * blockDim.x + tx;

	//to esure that  the extra threads do not do any work
	if( i < nRows && j < nCols) return;

		// Envolve computation
		// TODO: count how many neighbors are alive
		int nAliveNeig = 0;

		// index --> i * nCols + j
		
		//calculate the neighbors OH MADONNA GIà è SOLO PI§ disordinato xdxdxdxd
		int top_left =    compute_neighbor(i-1, j-1, nRows, nCols);
		int left = 		  compute_neighbor(i, j-1, nRows, nCols);
		int bottom_left = compute_neighbor(i+1, j-1, nRows, nCols);
		int top = 		  compute_neighbor(i-1, j, nRows, nCols);
		int top_right =   compute_neighbor(i-1, j+1, nRows, nCols);
		int right =       compute_neighbor(i, j+1, nRows, nCols);
		int bottom_right= compute_neighbor(i+1, j+1, nRows, nCols);
		int bottom =      compute_neighbor(i+1, j, nRows, nCols);

		//calculate how many neighbors around 3x3 are alive
		nAliveNeig = curr_gen[top_left] + curr_gen[left] + curr_gen[bottom_left] \
					+	curr_gen[top] + curr_gen[top_right] + curr_gen[right] 	\ 
					+ 	curr_gen[bottom_right] + curr_gen[bottom];
		
		// store computation in next_gen
		next_gen[ i * nCols + j] = ( nAliveNeig == 3 || (nAliveNeig == 2 && curr_gen[ i * nCols + j]));
		

}





void game(int nRows, int nCols, int timestep, int block_size ){

	int z, x, y;
	struct timeval start, end;
	double tot_time = 0.;

	//TODO: allocation in CPU and initialization
	unsigned int ** curr_gen = allocate_empty_gen(nRows, nCols);
	unsigned int ** next_gen = allocate_empty_gen(nRows, nCols); 
	
	//srand(10);
	for (x = 0; x < w; x++) for (y = 0; y < h; y++) curr_gen[y][x] = rand() < RAND_MAX / 10 ? ALIVE : DEAD;

	//TODO: allocation in GPU
	size_t gen_size = nRows * nCols * sizeof(unsigned int);

	unsigned int * cuda_curr_gen;
	unsigned int *cuda_next_gen;

	hipMalloc((void ** ) &cuda_curr_gen, gen_size );
	hipMalloc((void ** ) &cuda_next_gen, gen_size );

	// copy matrix from the host (CPU) to the device (GPU)
	hipMemcpy(cuda_curr_gen, curr_gen, gen_size, hipMemcpyHostToDevice);
	//hipMemset(cuda_next_gen, DEAD, gen_size); inutile secondo me TODO: vedere se eliminare sta riga

	//calculate how many block and how many thread per block
	
	dim3 block(block_size, block_size), dimGrid;
    dimGrid.x = ( nCols + block.x - 1)/block.x;
    dimGrid.y = ( nRows + block.y - 1)/block.y;
	



	for(z=0; z < timestep; z++){
			
			// get starting time at iteration z
			gettimeofday(&start, NULL);

			// Call Kernel on GPU
			cuda_envolve << dimGrid, block >> (cuda_curr_gen, cuda_next_gen, nRows, nCols);

			hipDeviceSynchronize(); 

			//swap cur_gen and next_gen
			swap(&cuda_curr_gen, &cuda_next_gen);

			// get ending time of iteration z
			gettimeofday(&end, NULL);
		
			// sum up the total time execution
			tot_time += (double) elapsed_wtime(start, end);
			
		
	}

	//free GPU memory
	hipFree(cuda_curr_gen);
	hipFree(cuda_next_gen);

	//free CPU memory
	free_gen(curr_gen);
	free_gen(next_gen);
}


  
 
int main(int c, char **v) {
	int w = 0, h = 0, t = 0, block_size = 32;

	if (c > 1) w = atoi(v[1]);
	if (c > 2) h = atoi(v[2]);
	if (c > 3) t = atoi(v[3]);
	if (c > 4) block_size = atoi(v[4]);

	if (w <= 0) w = 30;
	if (h <= 0) h = 30;
	if (t <= 0) t = 100;
	if (block_size < 32) block_size = 32; // number of threads per block
	
	game(w, h, t, block_size);
}

